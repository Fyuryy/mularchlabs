/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
__global__ void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double *d_input, *d_output;
    hipMalloc((void**)&d_input, length*length*sizeof(double));
    hipMalloc((void**)&d_output, length*length*sizeof(double));

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    hipMemcpy(d_input, input, length, hipMemcpyHostToDevice);

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */

    dim3 dimBlock(16, 16);
    dim3 dimGrid((length + dimBlock.x -1) / dimBlock.x, (length + dimBlock.y -1) / dimBlock.y);

    for(int n = 0; n < iterations; n++){
        array_process<<<dimGrid, dimBlock>>>(d_input, d_output, length, iterations);

        double *temp = d_input;
        d_input = d_output;
        d_output = temp;
    }


    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */

    hipMemcpy(output, d_output, length, hipMemcpyDeviceToHost);

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree(d_input);
    hipFree(d_output);
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}