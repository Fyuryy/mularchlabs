/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define MAX_HEAT 1000
#define BLK_SIZE 16

using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for (int n = 0; n < (int)iterations; n++)
    {
        for (int i = 1; i < length - 1; i++)
        {
            for (int j = 1; j < length - 1; j++)
            {
                output[(i) * (length) + (j)] = (input[(i - 1) * (length) + (j - 1)] +
                                                input[(i - 1) * (length) + (j)] +
                                                input[(i - 1) * (length) + (j + 1)] +
                                                input[(i) * (length) + (j - 1)] +
                                                input[(i) * (length) + (j)] +
                                                input[(i) * (length) + (j + 1)] +
                                                input[(i + 1) * (length) + (j - 1)] +
                                                input[(i + 1) * (length) + (j)] +
                                                input[(i + 1) * (length) + (j + 1)]) /
                                               9;
            }
        }
        output[(length / 2 - 1) * length + (length / 2 - 1)] = 1000;
        output[(length / 2) * length + (length / 2 - 1)] = 1000;
        output[(length / 2 - 1) * length + (length / 2)] = 1000;
        output[(length / 2) * length + (length / 2)] = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

/*__global void kernel_shared_mem(double *input, double *output, size_t length)
{

    //*border threads can be ignored as borders are always 0.
    extern shared double sharedMem[BLK_SIZE + 2];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int index = (i * length) + j;

    int sharedIndex = threadIdx.x * blockDim.y + threadIdx.y;
    sharedMem[sharedIndex] = input[(i * length) + j];
    syncthreads();

    // Modify the computation to use shared memory
    int sharedI;
    int sharedJ;

    if (0 < i && i < length - 1 && 0 < j && j < length - 1)
    {
        // Use shared memory in calculations, adjust indices appropriately
        double sum = 0;
        for (int y = -1; y <= 1; y++)
        {
            for (int x = -1; x <= 1; x++)
            {
                // Calculate the correct index for the shared memory
                sharedI = threadIdx.x + x;
                sharedJ = threadIdx.y + y;
                int sharedIdx = sharedJ * blockDim.x + sharedI;
                sum += sharedMem[sharedIdx];
            }
        }
        output[index] = sum / 9;
    }

    output[(length / 2 - 1) * length + (length / 2 - 1)] = MAX_HEAT;
    output[(length / 2) * length + (length / 2 - 1)] = MAX_HEAT;
    output[(length / 2 - 1) * length + (length / 2)] = MAX_HEAT;
    output[(length / 2) * length + (length / 2)] = MAX_HEAT;
}
*/

__global__ void kernel(double *input, double *output, size_t length)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int index = (i * length) + j;

    //*border threads can be ignored as borders are always 0.
    if (0 < i && i < length - 1 && 0 < j && j < length - 1)
    {output[index] = (input[(i - 1) * (length) + (j - 1)] +
                         input[(i - 1) * (length) + (j)] +
                         input[(i - 1) * (length) + (j + 1)] +
                         input[(i) * (length) + (j - 1)] +
                         input[(i) * (length) + (j)] +
                         input[(i) * (length) + (j + 1)] +
                         input[(i + 1) * (length) + (j - 1)] +
                         input[(i + 1) * (length) + (j)] + input[(i + 1) * (length) + (j + 1)]) /
                        9;

        output[(length / 2 - 1) * length + (length / 2 - 1)] = MAX_HEAT;
        output[(length / 2) * length + (length / 2 - 1)] = MAX_HEAT;
        output[(length / 2 - 1) * length + (length / 2)] = MAX_HEAT;
        output[(length / 2) * length + (length / 2)] = MAX_HEAT;
    }
}

// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    // Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */

    double *d_input, *d_output;
    size_t size = length * length * sizeof(double);

    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, size);

    /* Copying array from host to device goes here */
    hipEventRecord(cpy_H2D_start);

    hipMemcpy((double *)d_input, (double *)input, size, hipMemcpyHostToDevice);

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);
    // Copy array from host to device
    /* GPU calculation goes here */

    // We organize the thread blocks into 2D arrays of threads.
    int blocks = ceil((double)length / BLK_SIZE);
    dim3 gridSize(blocks, blocks);
    dim3 blockSize(BLK_SIZE, BLK_SIZE);

    double *temp;
    hipEventRecord(comp_start);
    // size_t sharedMemSize = (blockSize.x+2) * (blockSize.y+2) * sizeof(double); // blockSize = 32 if 32x32 threads

    for (int i = 0; i < iterations; i++)
    {
        // kernel_shared_mem<<<gridSize, blockSize, sharedMemSize>>>(d_input, d_output, length);
        kernel<<<gridSize, blockSize>>>(d_input, d_output, length);
        // kernel_noif<<<gridSize, blockSize>>>(d_input, d_output, length);

        temp = d_input;
        d_input = d_output;
        d_output = temp;
    }

    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);
    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);
    /* Postprocessing goes here */
    hipFree(d_input);
    hipFree(d_output);
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout << "Host to Device MemCpy takes " << setprecision(4) << time / 1000 << "s" << endl;
    hipEventElapsedTime(&time, comp_start, comp_end);
    cout << "Computation takes " << setprecision(4) << time / 1000 << "s" << endl;
    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout << "Device to Host MemCpy takes " << setprecision(4) << time / 1000 << "s" << endl;
}