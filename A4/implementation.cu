/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define MAX_HEAT 1000

using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for (int n = 0; n < (int)iterations; n++)
    {
        for (int i = 1; i < length - 1; i++)
        {
            for (int j = 1; j < length - 1; j++)
            {
                output[(i) * (length) + (j)] = (input[(i - 1) * (length) + (j - 1)] +
                                                input[(i - 1) * (length) + (j)] +
                                                input[(i - 1) * (length) + (j + 1)] +
                                                input[(i) * (length) + (j - 1)] +
                                                input[(i) * (length) + (j)] +
                                                input[(i) * (length) + (j + 1)] +
                                                input[(i + 1) * (length) + (j - 1)] +
                                                input[(i + 1) * (length) + (j)] +
                                                input[(i + 1) * (length) + (j + 1)]) /
                                               9;
            }
        }
        output[(length / 2 - 1) * length + (length / 2 - 1)] = 1000;
        output[(length / 2) * length + (length / 2 - 1)] = 1000;
        output[(length / 2 - 1) * length + (length / 2)] = 1000;
        output[(length / 2) * length + (length / 2)] = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

_global_ void kernel(double *input, double *output, size_t length)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int array_index = (j * ROW_SIZE) + i;
}

// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    // Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double *d_input, *d_output;
    size_t size = length * length * sizeof(double);

    if (!hipMalloc((void **)&d_input, size))
        return;

    if (hipMalloc((void **)&d_output, size))
        hipFree(d_input);
    return;

    hipEventRecord(cpy_H2D_start);

    /* Copying array from host to device goes here */
    if(!hipMemcpy(d_input, input, length, hipMemcpyHostToDevice));

    hipFree(d_input);
    hipFree(d_output);
    return;

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);
    // Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    dim3 thrsPerBlock(3, 4); // 3x4
    dim3 nBlks(2, 3);        // 2x3
    kernel<<<nBlks, thrsPerBlock>>>();
    //! must invoke our kernels here
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);
    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipMemcpy(output, d_output, length, hipMemcpyDeviceToHost);
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);
    /* Postprocessing goes here */
    hipFree(d_input);
    hipFree(d_output);
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout << "Host to Device MemCpy takes " << setprecision(4) << time / 1000 << "s" << endl;
    hipEventElapsedTime(&time, comp_start, comp_end);
    cout << "Computation takes " << setprecision(4) << time / 1000 << "s" << endl;
    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout << "Device to Host MemCpy takes " << setprecision(4) << time / 1000 << "s" << endl;
}